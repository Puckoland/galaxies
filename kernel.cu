#include "hip/hip_runtime.h"
#define BLOCK_DIM 2
#define BLOCK_SIZE BLOCK_DIM * BLOCK_DIM

__global__ void solve(sGalaxy A, sGalaxy B, float* distances, int n, int per_thread) {
//    printf("GRID: %d %d\n", gridDim.x, gridDim.y);
//    printf("PER: %d\n", per_thread);
    int base = BLOCK_SIZE * blockIdx.x;
    int bid = threadIdx.y * blockDim.x + threadIdx.x;
    int i = base + bid;

    __shared__ float D[BLOCK_SIZE];
    D[bid] = 0.0f;

    // TODO: load and compute in loops
    for (int t = 0; t < per_thread; t++) {
        int index = t * BLOCK_SIZE + i;

        __shared__ float As[BLOCK_SIZE * 3];
        __shared__ float Bs[BLOCK_SIZE * 3];

        // LOAD SHARED
        if (bid < n) {
            As[bid * 3] = A.x[index];
            As[bid * 3 + 1] = A.y[index];
            As[bid * 3 + 2] = A.z[index];

            Bs[bid * 3] = B.x[index];
            Bs[bid * 3 + 1] = B.y[index];
            Bs[bid * 3 + 2] = B.z[index];
        }
        __syncthreads();

        // PRINT SHARED MEMORY
//        if (bid == 0 && blockIdx.x == 0) {
//            printf("A:\n");
//            printf("%f %f %f\n", A.x[0], A.y[0], A.z[0]);
//            printf("%f %f %f\n", A.x[1], A.y[1], A.z[1]);
//            printf("%f %f %f\n", A.x[2], A.y[2], A.z[2]);
//            printf("%f %f %f\n", A.x[3], A.y[3], A.z[3]);
//            printf("As:\n");
//            printf("%f %f %f\n", As[0], As[1], As[2]);
//            printf("%f %f %f\n", As[3], As[4], As[5]);
//            printf("%f %f %f\n", As[6], As[7], As[8]);
//            printf("%f %f %f\n", As[9], As[10], As[11]);
//        }
//        if (bid == 0 && blockIdx.x == 1) {
//            printf("A:\n");
//            printf("%f %f %f\n", A.x[4], A.y[4], A.z[4]);
//            printf("As:\n");
//            printf("%f %f %f\n", As[0], As[1], As[2]);
//        }

        // COMPUTE
        float tmp = 0.0f;
        for (int k = bid + 1; k < BLOCK_SIZE && base + k < n; k++) {
            printf("%d %d\n", i, k);

            float Ax = A.x[i];
            float Ay = A.y[i];
            float Az = A.z[i];
            float Bx = B.x[i];
            float By = B.y[i];
            float Bz = B.z[i];

            if (blockIdx.x == 1) {
                printf("%f %f %f\n", Ax, Ay, Az);
                printf("%f %f %f\n", A.x[i], A.y[i], A.z[i]);
            }

            float da = sqrt(
                    (Ax - As[k * 3]) * (Ax - As[k * 3]) +
                    (Ay - As[k * 3 + 1]) * (Ay - As[k * 3 + 1]) +
                    (Az - As[k * 3 + 2]) * (Az - As[k * 3 + 2])
            );
            float db = sqrt(
                    (Bx - Bs[k * 3]) * (Bx - Bs[k * 3]) +
                    (By - Bs[k * 3 + 1]) * (By - Bs[k * 3 + 1]) +
                    (Bz - Bs[k * 3 + 2]) * (Bz - Bs[k * 3 + 2])
            );
            tmp += (da - db) * (da - db);
            //        printf("%d = %f\n", i, tmp);
            //        printf("(%d %d) D[%d] = %f\n", i, k, bid, tmp);
        }
        D[bid] += tmp;
    }
    __syncthreads();

    // REDUCE
    if (bid == 0) {
        float sum = 0.0f;
        for (int o = 0; o < BLOCK_SIZE; o++) {
            sum += D[o];
        }
//        printf("SUM (%d %d): %f\n", blockIdx.x, blockIdx.y, sum);
        atomicAdd(distances, sum);
    }
}

int roundUp(int value, int div) {
    if (value % div == 0) {
        return value / div;
    }
    return value / div + 1;
}

float solveGPU(sGalaxy A, sGalaxy B, int n) {
    int per_thread = roundUp(n, BLOCK_SIZE);
    int a = roundUp(n, BLOCK_SIZE);
    int b = 1;
    dim3 dimGrid (a, b);
    dim3 dimBlock (BLOCK_DIM, BLOCK_DIM);
    float* distances;
    size_t size = sizeof(*distances);
    hipMalloc(&distances, size);
    hipMemset(distances, 0, size);
    float* dist = (float *) malloc(size);
    if (dist == NULL) {
        fprintf(stderr, "Malloc failed");
        exit(1);
    }

    solve<<<dimGrid, dimBlock>>>(A, B, distances, n, per_thread);
    hipMemcpy(dist, distances, size, hipMemcpyDeviceToHost);

    float result = sqrt(1/((float)n*((float)n-1)) * *dist);
    free(dist);
    hipFree(distances);
    return result;
}
